#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>


long long cantidadIntervalos = 1000000000; // 1 B
double baseIntervalo;

__global__ void kernel(double* d_a, long long total_threads, double baseIntervalo, long long cantidadIntervalos)
{
	//calculate global thread ID(tid)
	long long tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	double acum = 0;
	double fdx = 0;
	double x = 0;

	if (tid < total_threads)
	{
		for (long long i = tid; i < cantidadIntervalos; i += total_threads)
		{
			x = (i+0.5) * baseIntervalo;
			fdx = 4 / (1 + x * x);
			acum += fdx;
		}
		acum *= baseIntervalo;
		d_a[tid] = acum;
	}
}

int main(void)
{
	//clock_t start, end;
	struct timespec start, end;

	hipError_t cudaStatus;
	hipDeviceProp_t prop;

	baseIntervalo = 1.0 / (double)cantidadIntervalos;
	double totalSum = 0;

	//Declare variables
	hipGetDeviceProperties(&prop, 0);
	int num_threads_supported = prop.maxThreadsPerBlock;
	int num_blocks_supported = prop.maxThreadsDim[0];
	
	
	//Grid Size
		//int NUM_BLOCKS = (int)ceil(datos / NUM_THREADS);
	int NUM_BLOCKS = num_blocks_supported;

	//Threadblock size
	int NUM_THREADS = num_threads_supported;
	long long total_threads = NUM_BLOCKS * NUM_THREADS;

	double* arr;
	//int size = total_threads * sizeof(double);

	//dynamic allocation
	hipMallocManaged(&arr, total_threads * sizeof(double));


	if (timespec_get(&start, TIME_UTC) != TIME_UTC)
	{
		printf("Error in calling timespec_get");
		exit(EXIT_FAILURE);
	}
	//Launch the kernel
	kernel << < NUM_BLOCKS, NUM_THREADS >> > (arr, total_threads, baseIntervalo, cantidadIntervalos);
	if (timespec_get(&end, TIME_UTC) != TIME_UTC)
	{
		printf("Error in calling timespec_get");
		exit(EXIT_FAILURE);
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//hipDeviceSynchronize waits for the kernel to finish, and returns
   // any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}



	for (long long c = 0; c < total_threads; c++)
	{
		totalSum += arr[c];
	}

Error:
	//De-allocate memory
	hipFree(arr);

	double total = (double)(end.tv_sec - start.tv_sec) + ((double)(end.tv_nsec - start.tv_nsec) / 1000000000L);
	printf("Result = %20.18lf (%.10lf ms)\n\n", totalSum, total);

	return 0;
}
